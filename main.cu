#include <hip/hip_runtime.h>
#include <iostream>

__global__ void add(int *a, int *b, int *c) {
    int index = threadIdx.x;
    c[index] = a[index] + b[index];
}

int main() {
    const int arraySize = 5;
    const int a[arraySize] = {1, 2, 3, 4, 5};
    const int b[arraySize] = {10, 20, 30, 40, 50};
    int c[arraySize] = {0};

    int *dev_a, *dev_b, *dev_c;

    hipMalloc((void**)&dev_a, arraySize * sizeof(int));
    hipMalloc((void**)&dev_b, arraySize * sizeof(int));
    hipMalloc((void**)&dev_c, arraySize * sizeof(int));

    hipMemcpy(dev_a, a, arraySize * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, arraySize * sizeof(int), hipMemcpyHostToDevice);

    add<<<1, arraySize>>>(dev_a, dev_b, dev_c);

    hipMemcpy(c, dev_c, arraySize * sizeof(int), hipMemcpyDeviceToHost);

    std::cout << "Result: ";
    for (int i = 0; i < arraySize; ++i) {
        std::cout << c[i] << " ";
    }
    std::cout << std::endl;

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}
